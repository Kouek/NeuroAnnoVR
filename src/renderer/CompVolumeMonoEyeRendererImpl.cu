#include "hip/hip_runtime.h"
#include "CompVolumeMonoEyeRendererImpl.h"

#include <Common/cuda_utils.hpp>

#include <>
#include <cuda_gl_interop.h>

using namespace kouek::CompVolumeRendererCUDA;

// CUDA Resource:
//   Allocated when needed,
//   freeed when CompVolumeRendererCUDA::MonoEyeFunc is deconstructed
__constant__ CompVolumeParameter dc_compVolumeParam;
__constant__ MonoEyeRenderParameter dc_renderParam;

__constant__ uint32_t dc_blockOffsets[MAX_LOD + 1];
__constant__ hipTextureObject_t dc_textures[MAX_TEX_UNIT_NUM];

__constant__ hipTextureObject_t dc_transferFunc;

hipArray_t d_preIntTFArray = nullptr;
hipTextureObject_t d_preIntTF;
__constant__ hipTextureObject_t dc_preIntTransferFunc;

uint32_t* d_mappingTable = nullptr;
__constant__ glm::uvec4* d_mappingTableStride4 = nullptr;

hipGraphicsResource_t outColorTexRsc = nullptr, inDepthTexRsc = nullptr;
glm::u8vec4* d_color = nullptr;
struct
{
	hipResourceDesc rscDesc;
	hipTextureDesc texDesc;
	hipTextureObject_t tex;
}d_depth;
size_t d_colorSize;
hipArray_t d_colorArr = nullptr, d_depthArr = nullptr;
hipStream_t stream = nullptr;

kouek::CompVolumeRendererCUDA::MonoEyeFunc::~MonoEyeFunc()
{
	if (d_preIntTFArray != nullptr)
	{
		CUDA_RUNTIME_CHECK(
			hipDestroyTextureObject(d_preIntTF));
		CUDA_RUNTIME_CHECK(
			hipFreeArray(d_preIntTFArray));
		d_preIntTFArray = nullptr;
	}
	// TODO
}

void kouek::CompVolumeRendererCUDA::MonoEyeFunc::uploadCompVolumeParam(const CompVolumeParameter& param)
{
	CUDA_RUNTIME_CHECK(
		hipMemcpyToSymbol(HIP_SYMBOL(dc_compVolumeParam), &param, sizeof(CompVolumeParameter)));
}

void kouek::CompVolumeRendererCUDA::MonoEyeFunc::uploadRenderParam(const MonoEyeRenderParameter& param)
{
	CUDA_RUNTIME_CHECK(
		hipMemcpyToSymbol(HIP_SYMBOL(dc_renderParam), &param, sizeof(MonoEyeRenderParameter)));
}

void kouek::CompVolumeRendererCUDA::MonoEyeFunc::uploadBlockOffs(const uint32_t* hostMemDat, size_t num)
{
	assert(num <= MAX_LOD + 1);
	CUDA_RUNTIME_CHECK(
		hipMemcpyToSymbol(HIP_SYMBOL(dc_blockOffsets), hostMemDat, sizeof(uint32_t) * num));
}

void kouek::CompVolumeRendererCUDA::MonoEyeFunc::uploadCUDATextureObj(const hipTextureObject_t* hostMemDat, size_t num)
{
	assert(num <= MAX_TEX_UNIT_NUM);
	CUDA_RUNTIME_CHECK(
		hipMemcpyToSymbol(HIP_SYMBOL(dc_textures), hostMemDat, sizeof(hipTextureObject_t) * num));
}

void kouek::CompVolumeRendererCUDA::MonoEyeFunc::uploadTransferFunc(const float* hostMemDat)
{
	// TODO
}

void kouek::CompVolumeRendererCUDA::MonoEyeFunc::uploadPreIntTransferFunc(const float* hostMemDat)
{
	if (d_preIntTFArray == nullptr)
		CreateCUDATexture2D(256, 256, &d_preIntTFArray, &d_preIntTF);
	UpdateCUDATexture2D(
		(uint8_t*)hostMemDat, d_preIntTFArray, sizeof(float) * 256 * 4, 256, 0, 0);
	CUDA_RUNTIME_CHECK(
		hipMemcpyToSymbol(HIP_SYMBOL(dc_preIntTransferFunc), &d_preIntTF, sizeof(hipTextureObject_t)));
}

void kouek::CompVolumeRendererCUDA::MonoEyeFunc::uploadMappingTable(const uint32_t* hostMemDat, size_t size)
{
	if (d_mappingTable == nullptr)
	{
		hipMalloc(&d_mappingTable, size);
		// cpy uint32_t ptr to uint4 ptr
		CUDA_RUNTIME_API_CALL(
			hipMemcpyToSymbol(HIP_SYMBOL(d_mappingTableStride4), &d_mappingTable, sizeof(glm::uvec4*)));
	}
	CUDA_RUNTIME_API_CALL(
		hipMemcpy(d_mappingTable, hostMemDat, size, hipMemcpyHostToDevice));
}

void kouek::CompVolumeRendererCUDA::MonoEyeFunc::registerGLResource(GLuint outColorTex, GLuint inDepthTex, uint32_t w, uint32_t h)
{
	d_colorSize = sizeof(glm::u8vec4) * w * h;
	CUDA_RUNTIME_API_CALL(
		hipGraphicsGLRegisterImage(&outColorTexRsc, outColorTex,
			GL_TEXTURE_2D, hipGraphicsRegisterFlagsWriteDiscard));
	CUDA_RUNTIME_API_CALL(
		hipMalloc(&d_color, d_colorSize));

	CUDA_RUNTIME_API_CALL(
		hipGraphicsGLRegisterImage(&inDepthTexRsc, inDepthTex,
			GL_TEXTURE_2D, hipGraphicsRegisterFlagsReadOnly));
	memset(&d_depth.rscDesc, 0, sizeof(hipResourceDesc));
	d_depth.rscDesc.resType = hipResourceTypeArray;
	memset(&d_depth.texDesc, 0, sizeof(hipTextureDesc));
	d_depth.texDesc.normalizedCoords = 0;
	d_depth.texDesc.filterMode = hipFilterModePoint;
	d_depth.texDesc.addressMode[0] = hipAddressModeClamp;
	d_depth.texDesc.addressMode[1] = hipAddressModeClamp;
	d_depth.texDesc.readMode = hipReadModeElementType;
}

void kouek::CompVolumeRendererCUDA::MonoEyeFunc::unregisterGLResource()
{
	if (outColorTexRsc != nullptr)
	{
		CUDA_RUNTIME_API_CALL(hipGraphicsUnregisterResource(outColorTexRsc));
		outColorTexRsc = nullptr;
		CUDA_RUNTIME_API_CALL(hipFree(d_color));
		d_color = nullptr;

		CUDA_RUNTIME_API_CALL(hipGraphicsUnregisterResource(inDepthTexRsc));
		inDepthTexRsc = nullptr;
	}
}

__device__ float virtualSampleLOD0(const glm::vec3& samplePos)
{
	// sample pos in Voxel Space -> virtual sample Block idx
	glm::uvec3 vsBlockIdx =
		samplePos / (float)dc_compVolumeParam.noPaddingBlockLength;

	// virtual sample Block idx -> real sample Block idx (in GPU Mem)
	glm::uvec4 GPUMemBlockIdx;
	{
		size_t flatVSBlockIdx = dc_blockOffsets[0]
			+ vsBlockIdx.z * dc_compVolumeParam.LOD0BlockDim.y * dc_compVolumeParam.LOD0BlockDim.x
			+ vsBlockIdx.y * dc_compVolumeParam.LOD0BlockDim.x
			+ vsBlockIdx.x;
		GPUMemBlockIdx = d_mappingTableStride4[flatVSBlockIdx];
	}

	if (((GPUMemBlockIdx.w >> 16) & (0x0000ffff)) != 1)
		// not a valid GPU Mem block
		return 0;

	// sample pos in Voxel Space -> real sample pos (in GPU Mem)
	glm::vec3 GPUMemSamplePos;
	{
		glm::vec3 offsetInNoPaddingBlock = samplePos -
			glm::vec3{ vsBlockIdx * dc_compVolumeParam.noPaddingBlockLength };
		GPUMemSamplePos = glm::vec3{ GPUMemBlockIdx.x, GPUMemBlockIdx.y, GPUMemBlockIdx.z }
			*(float)dc_compVolumeParam.blockLength
			+ offsetInNoPaddingBlock + (float)dc_compVolumeParam.padding;
		// normolized
		GPUMemSamplePos /= dc_renderParam.texUnitDim;
	}

	return tex3D<float>(dc_textures[GPUMemBlockIdx.w & (0x0000ffff)],
		GPUMemSamplePos.x, GPUMemSamplePos.y, GPUMemSamplePos.z);
}

__device__ glm::u8vec4 rgbaFloatToUbyte4(float r, float g, float b, float a)
{
	r = __saturatef(r); // clamp to [0.0, 1.0]
	g = __saturatef(g);
	b = __saturatef(b);
	a = __saturatef(a);
	r *= 255.f;
	g *= 255.f;
	b *= 255.f;
	a *= 255.f;
	return glm::u8vec4(r, g, b, a);
}

__device__ void rayIntersectAABB(
	float* tEnter, float* tExit,
	const glm::vec3& rayOri, const glm::vec3& rayDrc,
	const glm::vec3& bot, const glm::vec3& top)
{
	// For  Ori + Drc * t3Bot.x = <Bot.x, 0, 0>
	// Thus t3Bot.x = Bot.x / Drc.x
	// Thus t3Bot.y = Bot.x / Drc.y
	// If  \
			//  \_\|\ 
			//   \_\|
			//      \.t3Bot.x
			//      |\
			//    __|_\.___|
			//      |  \t3Bot.y
			//    __|___\._|_
			//    t3Top.y\ |
			//      |     \.t3Top.x
			// 
			// Then t3Min = t3Bot, t3Max = t3Top
			// And  the max of t3Min is tEnter
			// And  the min of t3Max is tExit

	glm::vec3 invRay = 1.f / rayDrc;
	glm::vec3 t3Bot = invRay * (bot - rayOri);
	glm::vec3 t3Top = invRay * (top - rayOri);
	glm::vec3 t3Min{
		fminf(t3Bot.x, t3Top.x),
		fminf(t3Bot.y, t3Top.y),
		fminf(t3Bot.z, t3Top.z) };
	glm::vec3 t3Max{
		fmaxf(t3Bot.x, t3Top.x),
		fmaxf(t3Bot.y, t3Top.y),
		fmaxf(t3Bot.z, t3Top.z) };
	*tEnter = fmaxf(fmaxf(t3Min.x, t3Min.y), fmaxf(t3Min.x, t3Min.z));
	*tExit = fminf(fminf(t3Max.x, t3Max.y), fminf(t3Max.x, t3Max.z));
}

__device__ glm::vec3 phongShadingLOD0(
	const glm::vec3& rayDrc, const glm::vec3& samplePos,
	const glm::vec3& diffuseColor)
{
	glm::vec3 N;
	{
		float val1, val2;
		val1 = virtualSampleLOD0(samplePos + glm::vec3{ 1.f,0,0 });
		val2 = virtualSampleLOD0(samplePos - glm::vec3{ 1.f,0,0 });
		N.x = val2 - val1;
		val1 = virtualSampleLOD0(samplePos + glm::vec3{ 0,1.f,0 });
		val2 = virtualSampleLOD0(samplePos - glm::vec3{ 0,1.f,0 });
		N.y = val2 - val1;
		val1 = virtualSampleLOD0(samplePos + glm::vec3{ 0,0,1.f });
		val2 = virtualSampleLOD0(samplePos - glm::vec3{ 0,0,1.f });
		N.z = val2 - val1;
	}
	N = glm::normalize(N);

	glm::vec3 L = { -rayDrc.x,-rayDrc.y,-rayDrc.z };
	glm::vec3 R = L;
	if (glm::dot(N, L) < 0) N = -N;

	glm::vec3 ambient = dc_renderParam.lightParam.ka * diffuseColor;
	glm::vec3 specular = glm::vec3(dc_renderParam.lightParam.ks
		* powf(fmaxf(dot(N, (L + R) / 2.f), 0),
			dc_renderParam.lightParam.shininess));
	glm::vec3 diffuse = dc_renderParam.lightParam.kd
		* fmaxf(dot(N, L), 0.f) * diffuseColor;

	return ambient + specular + diffuse;
}

// WARNING:
// - Declaring type of param d_depth as [const hipTextureObject_t &]
//   will cause unknown error at tex2D()
__global__ void renderKernel(glm::u8vec4* d_color, hipTextureObject_t d_depthTex)
{
	uint32_t windowX = blockIdx.x * blockDim.x + threadIdx.x;
	uint32_t windowY = blockIdx.y * blockDim.y + threadIdx.y;
	if (windowX >= dc_renderParam.windowSize.x || windowY >= dc_renderParam.windowSize.y) return;
	size_t windowFlatIdx = (size_t)windowY * dc_renderParam.windowSize.x + windowX;

	d_color[windowFlatIdx] = rgbaFloatToUbyte4(
		dc_renderParam.lightParam.bkgrndColor.r,
		dc_renderParam.lightParam.bkgrndColor.g,
		dc_renderParam.lightParam.bkgrndColor.b,
		dc_renderParam.lightParam.bkgrndColor.a);

	glm::vec3 rayDrc;
	float tEnter, tExit;
	{
		// find Ray of each Pixel on Window
		//   unproject
		glm::vec4 v41 = dc_renderParam.unProjection * glm::vec4{
			(((float)windowX / dc_renderParam.windowSize.x) - .5f) * 2.f,
			(((float)windowY / dc_renderParam.windowSize.y) - .5f) * 2.f,
			1.f, 1.f };
		//   don't rotate first to compute the Near&Far-clip steps
		rayDrc.x = v41.x, rayDrc.y = v41.y, rayDrc.z = v41.z;
		rayDrc = glm::normalize(rayDrc);
		float absRayDrcZ = fabsf(rayDrc.z);
		float tNearClip = dc_renderParam.nearClip / absRayDrcZ;
		float tFarClip = dc_renderParam.farClip;
		//   then compute upper bound of steps
		//   for Mesh-Volume mixed rendering
		{
			uchar4 depth4 = tex2D<uchar4>(d_depthTex, windowX, windowY);
			float meshBoundDep = dc_renderParam.projection23 /
				((depth4.x / 255.f * 2.f - 1.f) + dc_renderParam.projection22);
			if (tFarClip > meshBoundDep)
				tFarClip = meshBoundDep;
		}
		tFarClip /= absRayDrcZ;
		//   rotate
		v41.x = rayDrc.x, v41.y = rayDrc.y, v41.z = rayDrc.z; // normalized in vec3
		v41 = dc_renderParam.camRotaion * v41;
		rayDrc.x = v41.x, rayDrc.y = v41.y, rayDrc.z = v41.z;

		// Ray intersect Subregion(OBB)
		// equivalent to Ray intersect AABB in Subreion Space
		//   for pos, apply Rotation and Translation
		glm::vec4 v42{ dc_renderParam.camPos.x, dc_renderParam.camPos.y,
			dc_renderParam.camPos.z, 1.f };
		v42 = dc_renderParam.subrgn.fromWorldToSubrgn * v42;
		//   for drc, apply Rotation only
		v41.w = 0;
		v41 = dc_renderParam.subrgn.fromWorldToSubrgn * v41;
		rayIntersectAABB(
			&tEnter, &tExit,
			glm::vec3(v42),
			glm::normalize(glm::vec3(v41)),
			glm::zero<glm::vec3>(),
			glm::vec3{
				dc_renderParam.subrgn.halfW * 2,
				dc_renderParam.subrgn.halfH * 2,
				dc_renderParam.subrgn.halfD * 2 });

		// Near&Far-clip
		if (tEnter < tNearClip) tEnter = tNearClip;
		if (tExit > tFarClip) tExit = tFarClip;
	}

#ifdef TEST_RAY_DIRECTION
	// TEST: Ray Direction
	d_color[windowFlatIdx] = rgbaFloatToUbyte4(rayDrc.x, rayDrc.y, rayDrc.z, 1.f);
	return;
#endif // TEST_RAY_DIRECTION

	// no intersection
	if (tEnter >= tExit)
		return;
	glm::vec3 rayPos = dc_renderParam.camPos + tEnter * rayDrc;

#ifdef TEST_RAY_ENTER_EXIT_DIFF
	// TEST: Ray Enter Difference
	float diff = tExit - tEnter;
	d_color[windowFlatIdx] = rgbaFloatToUbyte4(diff, diff, diff, 1.f);
	return;
#endif // TEST_RAY_ENTER_EXIT_DIFF

#ifdef TEST_RAY_ENTER_POSITION
	// TEST: Ray Enter Position
	d_color[windowFlatIdx] = rgbaFloatToUbyte4(
		.5f * rayPos.x / d_renderParam.subrgn.halfW,
		.5f * rayPos.y / d_renderParam.subrgn.halfH,
		.5f * rayPos.z / d_renderParam.subrgn.halfD, 1.f);
	return;
#endif // TEST_RAY_ENTER_POSITION

#ifdef TEST_RAY_EXIT_POSITION
	// TEST: Ray Exit Position
	rayPos = d_renderParam.camPos + tExit * rayDrc;
	d_color[windowFlatIdx] = rgbaFloatToUbyte4(
		.5f * rayPos.x / d_renderParam.subrgn.halfW,
		.5f * rayPos.y / d_renderParam.subrgn.halfH,
		.5f * rayPos.z / d_renderParam.subrgn.halfD, 1.f);
	return;
#endif // TEST_RAY_EXIT_POSITION

	glm::vec3 subrgnCenterInWdSp = {
		.5f * dc_renderParam.subrgn.halfW,
		.5f * dc_renderParam.subrgn.halfH,
		.5f * dc_renderParam.subrgn.halfD,
	};
	glm::vec3 rayDrcMulStp = rayDrc * dc_renderParam.step;
	glm::vec3 samplePos;
	glm::vec4 color = glm::zero<glm::vec4>();
	float sampleVal = 0;
	uint32_t stepNum = 0;
	for (;
		stepNum <= dc_renderParam.maxStepNum && tEnter <= tExit;
		++stepNum, tEnter += dc_renderParam.step, rayPos += rayDrcMulStp)
	{
		// ray pos in World Space -> sample pos in Voxel Space
		samplePos =
			(rayPos - subrgnCenterInWdSp + dc_renderParam.subrgn.center)
			/ dc_compVolumeParam.spaces;

		// virtual sample in Voxel Space, real sample in GPU Mem
		float currSampleVal = virtualSampleLOD0(samplePos);
		if (currSampleVal <= 0)
			continue;

		float4 currColor = tex2D<float4>(dc_preIntTransferFunc, sampleVal, currSampleVal);
		if (currColor.w <= 0)
			continue;

		glm::vec3 shadingColor = phongShadingLOD0(rayDrc,
			samplePos, glm::vec3{ currColor.x,currColor.y,currColor.z });
		currColor.x = shadingColor.x;
		currColor.y = shadingColor.y;
		currColor.z = shadingColor.z;

		sampleVal = currSampleVal;
		color = color + (1.f - color.w) * glm::vec4{ currColor.x,currColor.y,currColor.z,currColor.w }
		*glm::vec4{ currColor.w,currColor.w,currColor.w,1.f };

		if (color.w > 0.9f)
			break;
	}

	// gamma correction
	constexpr float GAMMA_CORRECT_COEF = 1.f / 2.2f;
	color.r = powf(color.r, GAMMA_CORRECT_COEF);
	color.g = powf(color.g, GAMMA_CORRECT_COEF);
	color.b = powf(color.b, GAMMA_CORRECT_COEF);

	d_color[windowFlatIdx] = rgbaFloatToUbyte4(color.r, color.g, color.b, color.a);
}

void kouek::CompVolumeRendererCUDA::MonoEyeFunc::render(uint32_t windowW, uint32_t windowH)
{
	if (stream == nullptr)
		CUDA_RUNTIME_CHECK(hipStreamCreate(&stream));

	// from here, called per frame, thus no CUDA_RUNTIME_API_CHECK
	hipGraphicsMapResources(1, &outColorTexRsc, stream);
	hipGraphicsSubResourceGetMappedArray(&d_colorArr, outColorTexRsc, 0, 0);

	hipGraphicsMapResources(1, &inDepthTexRsc, stream);
	hipGraphicsSubResourceGetMappedArray(&d_depthArr, inDepthTexRsc, 0, 0);
	d_depth.rscDesc.res.array.array = d_depthArr;
	hipCreateTextureObject(&d_depth.tex, &d_depth.rscDesc,
		&d_depth.texDesc, nullptr);

	dim3 threadPerBlock = { 16, 16 };
	dim3 blockPerGrid = { (windowW + threadPerBlock.x - 1) / threadPerBlock.x,
						 (windowH + threadPerBlock.y - 1) / threadPerBlock.y };
	renderKernel <<< blockPerGrid, threadPerBlock, 0, stream >>> (d_color, d_depth.tex);

	hipMemcpyToArray(d_colorArr, 0, 0,
		d_color, d_colorSize, hipMemcpyDeviceToDevice);

	d_colorArr = d_depthArr = nullptr;
	hipGraphicsUnmapResources(1, &outColorTexRsc, stream);
	hipGraphicsUnmapResources(1, &inDepthTexRsc, stream);
}
