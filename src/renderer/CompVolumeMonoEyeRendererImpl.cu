#include "hip/hip_runtime.h"
#include "CompVolumeMonoEyeRendererImpl.h"

#include <Common/cuda_utils.hpp>

#include <>
#include <cuda_gl_interop.h>

namespace kouek
{
	namespace CompVolumeMonoEyeRendererImplCUDA
	{
		__constant__ uint32_t d_blockOffsets[MAX_LOD + 1];
		void uploadBlockOffs(
			const uint32_t* hostMemDat, size_t num)
		{
			assert(num <= MAX_LOD + 1);
			CUDA_RUNTIME_CHECK(
				hipMemcpyToSymbol(HIP_SYMBOL(d_blockOffsets), hostMemDat, sizeof(uint32_t) * num));
		}

		__constant__ CompVolumeParameter d_compVolumeParam;
		void uploadCompVolumeParam(const CompVolumeParameter* hostMemDat)
		{
			CUDA_RUNTIME_CHECK(
				hipMemcpyToSymbol(HIP_SYMBOL(d_compVolumeParam), hostMemDat, sizeof(CompVolumeParameter)));
		}

		__constant__ hipTextureObject_t d_textures[MAX_TEX_UNIT_NUM];
		void uploadCUDATextureObj(
			const hipTextureObject_t* hostMemDat, size_t num)
		{
			assert(num <= MAX_TEX_UNIT_NUM);
			CUDA_RUNTIME_CHECK(
				hipMemcpyToSymbol(HIP_SYMBOL(d_textures), hostMemDat, sizeof(hipTextureObject_t) * num));
		}

		__constant__ hipTextureObject_t d_transferFunc;
		void uploadTransferFunc(
			const float* hostMemDat)
		{
			// TODO
		}

		hipArray* preIntTFArray = nullptr;
		hipTextureObject_t preIntTF;
		__constant__ hipTextureObject_t d_preIntTransferFunc;
		void uploadPreIntTransferFunc(
			const float* hostMemDat)
		{
			if (preIntTFArray == nullptr)
				CreateCUDATexture2D(256, 256, &preIntTFArray, &preIntTF);
			UpdateCUDATexture2D(
				(uint8_t*)hostMemDat, preIntTFArray, sizeof(float) * 256 * 4, 256, 0, 0);
			CUDA_RUNTIME_CHECK(
				hipMemcpyToSymbol(HIP_SYMBOL(d_preIntTransferFunc), &preIntTF, sizeof(hipTextureObject_t)));
		}

		__constant__ RenderParameter d_renderParam;
		void uploadRenderParam(const RenderParameter* hostMemDat)
		{
			CUDA_RUNTIME_CHECK(
				hipMemcpyToSymbol(HIP_SYMBOL(d_renderParam), hostMemDat, sizeof(RenderParameter)));
		}

		uint32_t* d_mappingTable = nullptr;
		__constant__ uint4* d_mappingTableStride4 = nullptr;
		void uploadMappingTable(const uint32_t* hostMemDat, size_t size)
		{
			if (d_mappingTable == nullptr)
			{
				hipMalloc(&d_mappingTable, size);
				// cpy uint32_t ptr to uint4 ptr
				CUDA_RUNTIME_API_CALL(
					hipMemcpyToSymbol(HIP_SYMBOL(d_mappingTableStride4), &d_mappingTable, sizeof(uint4*)));
			}
			CUDA_RUNTIME_API_CALL(
				hipMemcpy(d_mappingTable, hostMemDat, size, hipMemcpyHostToDevice));
		}

		hipGraphicsResource_t PBORsc = nullptr;
		void registerOutputGLPBO(GLuint outPBO)
		{
			CUDA_RUNTIME_API_CALL(
				hipGraphicsGLRegisterBuffer(&PBORsc, outPBO, cudaGraphicsMapFlagsWriteDiscard));
		}

		void unregisterOutputGLPBO()
		{
			if (PBORsc != nullptr)
			{
				CUDA_RUNTIME_API_CALL(
					hipGraphicsUnregisterResource(PBORsc));
				PBORsc = nullptr;
			}
		}

		__device__ uint32_t rgbaFloatToUInt32(float r, float g, float b, float a)
		{
			r = __saturatef(r); // clamp to [0.0, 1.0]
			g = __saturatef(g);
			b = __saturatef(b);
			a = __saturatef(a);
			return (uint32_t(r * 255) << 24) | (uint32_t(g * 255) << 16)
				| (uint32_t(b * 255) << 8) | uint32_t(a * 255);
		}

		__device__ void rayIntersectAABB(
			float* tEnter, float* tExit,
			const glm::vec3& rayOri, const glm::vec3& rayDrc,
			const glm::vec3& bot, const glm::vec3& top)
		{
			// For  Ori + Drc * t3Bot.x = <Bot.x, 0, 0>
			// Thus t3Bot.x = Bot.x / Drc.x
			// Thus t3Bot.y = Bot.x / Drc.y
			// If
			//   _\|
			//     \
			//      \.t3Bot.x
			//      |\
			//    __|_\.___|
			//      |  \t3Bot.y
			//    __|___\._|_
			//    t3Top.y\ |
			//      |     \.t3Top.x
			// 
			// Then t3Min = t3Bot, t3Max = t3Top
			// And  the max of t3Min is tEnter
			// And  the min of t3Max is tExit

			glm::vec3 invRay = 1.f / rayDrc;
			glm::vec3 t3Bot = invRay * (bot - rayOri);
			glm::vec3 t3Top = invRay * (top - rayOri);
			glm::vec3 t3Min{
				fminf(t3Bot.x, t3Top.x),
				fminf(t3Bot.y, t3Top.y),
				fminf(t3Bot.z, t3Top.z) };
			glm::vec3 t3Max{
				fmaxf(t3Bot.x, t3Top.x),
				fmaxf(t3Bot.y, t3Top.y),
				fmaxf(t3Bot.z, t3Top.z) };
			*tEnter = fmaxf(fmaxf(t3Min.x, t3Min.y), fmaxf(t3Min.x, t3Min.z));
			*tExit = fminf(fminf(t3Max.x, t3Max.y), fminf(t3Max.x, t3Max.z));
		}

		__global__ void renderKernel(uint32_t* d_window)
		{
			uint32_t windowX = blockIdx.x * blockDim.x + threadIdx.x;
			uint32_t windowY = blockIdx.y * blockDim.y + threadIdx.y;
			if (windowX >= d_renderParam.windowSize.x || windowY >= d_renderParam.windowSize.y) return;
			size_t windowFlatIdx = (size_t)windowY * d_renderParam.windowSize.x + windowX;

			glm::vec3 rayDrc;
			float tEnter, tExit;
			{
				float offsX = (((float)windowX / d_renderParam.windowSize.x) - .5f) * 2.f;
				float offsY = (((float)windowY / d_renderParam.windowSize.y) - .5f) * 2.f;
				glm::vec4 v41 = d_renderParam.unProjection * glm::vec4(offsX, offsY, 1.f, 1.f);
				v41 = d_renderParam.camRotaion * v41;
				rayDrc.x = v41.x, rayDrc.y = v41.y, rayDrc.z = v41.z;
				rayDrc = glm::normalize(rayDrc);

				// Ray intersect Subregion(OBB)
				// equivalent to Ray intersect AABB in Subreion Space
				//   for pos, apply Rotation and Translation
				glm::vec4 v42{ d_renderParam.camPos.x, d_renderParam.camPos.y,
					d_renderParam.camPos.z, 1.f };
				v42 = d_renderParam.subrgn.fromWorldToSubrgn * v42;
				//   for drc, apply Rotation only
				v41.x = rayDrc.x, v41.y = rayDrc.y, v41.z = rayDrc.z, v41.w = 0;
				v41 = d_renderParam.subrgn.fromWorldToSubrgn * v41;
				rayIntersectAABB(
					&tEnter, &tExit,
					glm::vec3(v42), glm::normalize(glm::vec3(v41)),
					glm::zero<glm::vec3>(),
					glm::vec3{
						d_renderParam.subrgn.halfW * 2,
						d_renderParam.subrgn.halfH * 2,
						d_renderParam.subrgn.halfD * 2 });
			}

#ifdef TEST_RAY_DIRECTION
			// TEST: Ray Direction
			d_window[windowFlatIdx] = rgbaFloatToUInt32(rayDrc.x, rayDrc.y, rayDrc.z, 1.f);
			return;
#endif // TEST_RAY_DIRECTION

			if (tEnter < 0) tEnter = 0;
			if (tEnter >= tExit)
			{
				d_window[windowFlatIdx] = rgbaFloatToUInt32(
					d_renderParam.lightParam.bkgrndColor.r,
					d_renderParam.lightParam.bkgrndColor.g,
					d_renderParam.lightParam.bkgrndColor.b,
					d_renderParam.lightParam.bkgrndColor.a);
				return;
			}
			glm::vec3 rayPos = d_renderParam.camPos + tEnter * rayDrc;
#define TEST_RAY_ENTER_POSITION
#ifdef TEST_RAY_ENTER_POSITION
			// TEST: Ray Enter Position
			d_window[windowFlatIdx] = rgbaFloatToUInt32(
				.5f * rayPos.x / d_renderParam.subrgn.halfW,
				.5f * rayPos.y / d_renderParam.subrgn.halfH,
				.5f * rayPos.z / d_renderParam.subrgn.halfD, 1.f);
			return;
#endif // TEST_RAY_ENTER_POSITION
		}

		uint32_t* d_window = nullptr;
		hipStream_t stream = nullptr;
		void render(uint32_t windowW, uint32_t windowH)
		{
			if (stream == nullptr)
				CUDA_RUNTIME_CHECK(hipStreamCreate(&stream));

			// from here, called per frame, thus no CUDA_RUNTIME_API_CHECK
			hipGraphicsMapResources(1, &PBORsc, stream);
			size_t rscSize;
			hipGraphicsResourceGetMappedPointer((void**)&d_window, &rscSize, PBORsc);

			dim3 threadPerBlock = { 16, 16 };
			dim3 blockPerGrid = { (windowW + threadPerBlock.x - 1) / threadPerBlock.x,
								 (windowH + threadPerBlock.y - 1) / threadPerBlock.y };
			renderKernel<<<blockPerGrid, threadPerBlock, 0, stream >>>(d_window);

			hipGraphicsUnmapResources(1, &PBORsc, stream);
			d_window = nullptr;
		}
	}
}
