#include "hip/hip_runtime.h"
#include "CompVolumeFAVRRendererImpl.h"

#include <Common/cuda_utils.hpp>

#include <>
#include <cuda_gl_interop.h>

using namespace kouek::CompVolumeRendererCUDA;

// CUDA Resource:
//   Allocated when needed,
//   freeed when CompVolumeRendererCUDA::FAVRFunc is deconstructed
__constant__ CompVolumeParameter d_compVolumeParam;
__constant__ FAVRRenderParameter d_renderParam;

__constant__ uint32_t d_blockOffsets[MAX_LOD + 1];
__constant__ hipTextureObject_t d_textures[MAX_TEX_UNIT_NUM];

__constant__ hipTextureObject_t d_transferFunc;

hipArray_t d_preIntTFArray = nullptr;
hipTextureObject_t d_preIntTF;
__constant__ hipTextureObject_t d_preIntTransferFunc;

uint32_t* d_mappingTable = nullptr;
__constant__ glm::uvec4* d_mappingTableStride4 = nullptr;

hipArray_t d_sbssmplTexArrs[MAX_SUBSAMPLE_LEVEL_NUM] = { nullptr };
hipArray_t d_reconsTexArrs[MAX_SUBSAMPLE_LEVEL_NUM] = { nullptr };
hipSurfaceObject_t sbsmplTexes[MAX_SUBSAMPLE_LEVEL_NUM];
hipSurfaceObject_t reconsTexes[MAX_SUBSAMPLE_LEVEL_NUM];
__constant__ float d_sbsmplRadSqrs[MAX_SUBSAMPLE_LEVEL_NUM + 1] = { 0 };
__constant__ hipSurfaceObject_t d_sbsmplTexes[MAX_SUBSAMPLE_LEVEL_NUM];
__constant__ hipSurfaceObject_t d_reconsTexes[MAX_SUBSAMPLE_LEVEL_NUM];

hipGraphicsResource_t outColorTexRsc2[2] = { nullptr };
hipGraphicsResource_t inDepthTexRsc2[2] = { nullptr };
glm::u8vec4* d_color2[2] = { nullptr };
struct
{
	hipResourceDesc rscDesc;
	hipTextureDesc texDesc;
}depthTexDesc;
hipTextureObject_t d_depthTex2[2];
size_t d_colorSize;
hipArray_t d_colorArr2[2] = { nullptr }, d_depthArr2[2] = { nullptr };
hipStream_t stream = nullptr;

kouek::CompVolumeRendererCUDA::FAVRFunc::~FAVRFunc()
{
	if (d_preIntTFArray != nullptr)
	{
		CUDA_RUNTIME_CHECK(
			hipDestroyTextureObject(d_preIntTF));
		CUDA_RUNTIME_CHECK(
			hipFreeArray(d_preIntTFArray));
		d_preIntTFArray = nullptr;
	}
	// TODO
}

void kouek::CompVolumeRendererCUDA::FAVRFunc::uploadCompVolumeParam(const CompVolumeParameter& param)
{
	CUDA_RUNTIME_CHECK(
		hipMemcpyToSymbol(HIP_SYMBOL(d_compVolumeParam), &param, sizeof(CompVolumeParameter)));
}

void kouek::CompVolumeRendererCUDA::FAVRFunc::uploadRenderParam(
	const FAVRRenderParameter& param)
{
	CUDA_RUNTIME_CHECK(
		hipMemcpyToSymbol(HIP_SYMBOL(d_renderParam), &param, sizeof(FAVRRenderParameter)));
}

void kouek::CompVolumeRendererCUDA::FAVRFunc::uploadBlockOffs(const uint32_t* hostMemDat, size_t num)
{
	assert(num <= MAX_LOD + 1);
	CUDA_RUNTIME_CHECK(
		hipMemcpyToSymbol(HIP_SYMBOL(d_blockOffsets), hostMemDat, sizeof(uint32_t) * num));
}

void kouek::CompVolumeRendererCUDA::FAVRFunc::uploadCUDATextureObj(const hipTextureObject_t* hostMemDat, size_t num)
{
	assert(num <= MAX_TEX_UNIT_NUM);
	CUDA_RUNTIME_CHECK(
		hipMemcpyToSymbol(HIP_SYMBOL(d_textures), hostMemDat, sizeof(hipTextureObject_t) * num));
}

void kouek::CompVolumeRendererCUDA::FAVRFunc::uploadTransferFunc(const float* hostMemDat)
{
	// TODO
}

void kouek::CompVolumeRendererCUDA::FAVRFunc::uploadPreIntTransferFunc(const float* hostMemDat)
{
	if (d_preIntTFArray == nullptr)
		CreateCUDATexture2D(256, 256, &d_preIntTFArray, &d_preIntTF);
	UpdateCUDATexture2D(
		(uint8_t*)hostMemDat, d_preIntTFArray, sizeof(float) * 256 * 4, 256, 0, 0);
	CUDA_RUNTIME_CHECK(
		hipMemcpyToSymbol(HIP_SYMBOL(d_preIntTransferFunc), &d_preIntTF, sizeof(hipTextureObject_t)));
}

void kouek::CompVolumeRendererCUDA::FAVRFunc::uploadMappingTable(const uint32_t* hostMemDat, size_t size)
{
	if (d_mappingTable == nullptr)
	{
		hipMalloc(&d_mappingTable, size);
		// cpy uint32_t ptr to uint4 ptr
		CUDA_RUNTIME_API_CALL(
			hipMemcpyToSymbol(HIP_SYMBOL(d_mappingTableStride4), &d_mappingTable, sizeof(glm::uvec4*)));
	}
	CUDA_RUNTIME_API_CALL(
		hipMemcpy(d_mappingTable, hostMemDat, size, hipMemcpyHostToDevice));
}

void kouek::CompVolumeRendererCUDA::FAVRFunc::registerGLResource(
	GLuint outLftColorTex, GLuint outRhtColorTex,
	GLuint inLftDepthTex, GLuint inRhtDepthTex,
	uint32_t w, uint32_t h)
{
	d_colorSize = sizeof(glm::u8vec4) * w * h;
	CUDA_RUNTIME_API_CALL(
		hipGraphicsGLRegisterImage(&outColorTexRsc2[0], outLftColorTex,
			GL_TEXTURE_2D, hipGraphicsRegisterFlagsWriteDiscard));
	CUDA_RUNTIME_API_CALL(
		hipGraphicsGLRegisterImage(&outColorTexRsc2[1], outRhtColorTex,
			GL_TEXTURE_2D, hipGraphicsRegisterFlagsWriteDiscard));
	CUDA_RUNTIME_API_CALL(
		hipMalloc(&d_color2[0], d_colorSize));
	CUDA_RUNTIME_API_CALL(
		hipMalloc(&d_color2[1], d_colorSize));

	CUDA_RUNTIME_API_CALL(
		hipGraphicsGLRegisterImage(&inDepthTexRsc2[0], inLftDepthTex,
			GL_TEXTURE_2D, hipGraphicsRegisterFlagsReadOnly));
	CUDA_RUNTIME_API_CALL(
		hipGraphicsGLRegisterImage(&inDepthTexRsc2[1], inRhtDepthTex,
			GL_TEXTURE_2D, hipGraphicsRegisterFlagsReadOnly));

	memset(&depthTexDesc.rscDesc, 0, sizeof(hipResourceDesc));
	depthTexDesc.rscDesc.resType = hipResourceTypeArray;
	memset(&depthTexDesc.texDesc, 0, sizeof(hipTextureDesc));
	depthTexDesc.texDesc.normalizedCoords = 0;
	depthTexDesc.texDesc.filterMode = hipFilterModePoint;
	depthTexDesc.texDesc.addressMode[0] = hipAddressModeClamp;
	depthTexDesc.texDesc.addressMode[1] = hipAddressModeClamp;
	depthTexDesc.texDesc.readMode = hipReadModeElementType;
}

void kouek::CompVolumeRendererCUDA::FAVRFunc::unregisterGLResource()
{
	for (uint8_t idx = 0; idx < 2; ++idx)
		if (outColorTexRsc2[idx] != nullptr)
		{
			CUDA_RUNTIME_API_CALL(hipGraphicsUnregisterResource(outColorTexRsc2[idx]));
			outColorTexRsc2[idx] = nullptr;
			CUDA_RUNTIME_API_CALL(hipFree(d_color2[idx]));
			d_color2[idx] = nullptr;

			CUDA_RUNTIME_API_CALL(hipGraphicsUnregisterResource(inDepthTexRsc2[idx]));
			inDepthTexRsc2[idx] = nullptr;
		}
}

__global__ void createSubsampleSurfKernel(
	glm::vec4* d_sbsmpl,
	uint32_t sbssmplTexW, uint32_t sbssmplTexH)
{
	uint32_t texX = blockIdx.x * blockDim.x + threadIdx.x;
	uint32_t texY = blockIdx.y * blockDim.y + threadIdx.y;
	size_t texFlatIdx = texY * sbssmplTexW + texX;
	if (texX > sbssmplTexW|| texY > sbssmplTexH) return;

	float scale, invLvl = 1.f / d_renderParam.sbsmplLvl;
	float centerX, centerY, windowCenterX, windowCenterY;
	centerX = centerY = .5f * sbssmplTexW;
	windowCenterX = .5f * d_renderParam.windowSize.x;
	windowCenterY = .5f * d_renderParam.windowSize.y;
	glm::vec4 X_Y_HasVal__;
#pragma unroll
	for (uint8_t stage = 0; stage < d_renderParam.sbsmplLvl;
		++stage, centerY += sbssmplTexW)
	{
		float diffX = (float)texX - centerX, diffY = (float)texY - centerY;
		float radSqr = diffX * diffX + diffY * diffY;
		scale = 1.f - (float)stage * invLvl;
		if (radSqr >= d_sbsmplRadSqrs[stage] && radSqr < d_sbsmplRadSqrs[stage + 1])
			X_Y_HasVal__.r = X_Y_HasVal__.g = X_Y_HasVal__.b = X_Y_HasVal__.w = 0;
		else
		{
			diffX /= scale, diffY /= scale;
			diffX += windowCenterX, diffY += windowCenterY;
			X_Y_HasVal__.r = diffX / d_renderParam.windowSize.x;
			X_Y_HasVal__.g = diffY / d_renderParam.windowSize.y;
			X_Y_HasVal__.b = 0;
			X_Y_HasVal__.w = 1.f;
		}
		d_sbsmpl[texFlatIdx] = X_Y_HasVal__;
	}
}

static void createSubsampleAndReconsTexes(uint8_t lvl, uint32_t w, uint32_t h)
{
	// 0 has been assigned to d_sbsmplRadSqrs[0]
	for (uint8_t stage = 1; stage < lvl; ++stage)
	{
		float radSqr = (float)w / stage / 2.f;
		radSqr *= radSqr;
		CUDA_RUNTIME_API_CALL(hipMemcpyToSymbol(HIP_SYMBOL(
			d_sbsmplRadSqrs[stage]), &radSqr, sizeof(float)));
	}
	{
		float radSqr = INFINITY;
		CUDA_RUNTIME_API_CALL(hipMemcpyToSymbol(HIP_SYMBOL(
			d_sbsmplRadSqrs[lvl]), &radSqr, sizeof(float)));
	}
	uint8_t idx = lvl - 1;
	uint32_t sbssmplTexW = w / lvl;
	uint32_t sbssmplTexH = sbssmplTexW * lvl;
	{
		hipChannelFormatDesc chnnlDesc = hipCreateChannelDesc<float4>();
		CUDA_RUNTIME_API_CALL(
			hipMallocArray(&d_sbssmplTexArrs[idx], &chnnlDesc,
				sbssmplTexW, sbssmplTexH, hipArraySurfaceLoadStore));
	}
	{
		hipResourceDesc rscDesc;
		memset(&rscDesc, 0, sizeof(hipResourceDesc));
		rscDesc.resType = hipResourceTypeArray;
		rscDesc.res.array.array = d_sbssmplTexArrs[idx];
		hipTextureDesc texDesc;
		memset(&depthTexDesc.texDesc, 0, sizeof(hipTextureDesc));
		depthTexDesc.texDesc.normalizedCoords = 1;
		depthTexDesc.texDesc.filterMode = hipFilterModeLinear;
		depthTexDesc.texDesc.addressMode[0] = hipAddressModeClamp;
		depthTexDesc.texDesc.addressMode[1] = hipAddressModeClamp;
		depthTexDesc.texDesc.readMode = hipReadModeNormalizedFloat;
		CUDA_RUNTIME_API_CALL(
			hipCreateTextureObject(&sbsmplTexes[idx], &rscDesc, &texDesc, nullptr));
		CUDA_RUNTIME_API_CALL(hipMemcpyToSymbol(HIP_SYMBOL(
			d_sbsmplTexes[idx]), &sbsmplTexes[idx], sizeof(hipSurfaceObject_t)));
	}
	{
		glm::vec4* d_sbsmpl = nullptr;
		size_t d_sbsmplSize = sizeof(glm::vec4) * sbssmplTexW * sbssmplTexH;
		CUDA_RUNTIME_API_CALL(hipMalloc(&d_sbsmpl, d_sbsmplSize));

		dim3 threadPerBlock = { 16, 16 };
		dim3 blockPerGrid = { (sbssmplTexW + threadPerBlock.x - 1) / threadPerBlock.x,
							 (sbssmplTexH + threadPerBlock.y - 1) / threadPerBlock.y };
		createSubsampleSurfKernel << <blockPerGrid, threadPerBlock, 0, stream >> > (
			d_sbsmpl, sbssmplTexW, sbssmplTexH);

		CUDA_RUNTIME_API_CALL(hipMemcpyToArray(
			d_sbssmplTexArrs[idx], 0, 0, d_sbsmpl, d_sbsmplSize, hipMemcpyDeviceToDevice));
		CUDA_RUNTIME_API_CALL(hipFree(d_sbsmpl));
	}
}

__device__ float virtualSampleLOD0(const glm::vec3& samplePos)
{
	// sample pos in Voxel Space -> virtual sample Block idx
	glm::uvec3 vsBlockIdx =
		samplePos / (float)d_compVolumeParam.noPaddingBlockLength;

	// virtual sample Block idx -> real sample Block idx (in GPU Mem)
	glm::uvec4 GPUMemBlockIdx;
	{
		size_t flatVSBlockIdx = d_blockOffsets[0]
			+ vsBlockIdx.z * d_compVolumeParam.LOD0BlockDim.y * d_compVolumeParam.LOD0BlockDim.x
			+ vsBlockIdx.y * d_compVolumeParam.LOD0BlockDim.x
			+ vsBlockIdx.x;
		GPUMemBlockIdx = d_mappingTableStride4[flatVSBlockIdx];
	}

	if (((GPUMemBlockIdx.w >> 16) & (0x0000ffff)) != 1)
		// not a valid GPU Mem block
		return 0;

	// sample pos in Voxel Space -> real sample pos (in GPU Mem)
	glm::vec3 GPUMemSamplePos;
	{
		glm::vec3 offsetInNoPaddingBlock = samplePos -
			glm::vec3{ vsBlockIdx * d_compVolumeParam.noPaddingBlockLength };
		GPUMemSamplePos = glm::vec3{ GPUMemBlockIdx.x, GPUMemBlockIdx.y, GPUMemBlockIdx.z }
			*(float)d_compVolumeParam.blockLength
			+ offsetInNoPaddingBlock + (float)d_compVolumeParam.padding;
		// normolized
		GPUMemSamplePos /= d_renderParam.texUnitDim;
	}

	return tex3D<float>(d_textures[GPUMemBlockIdx.w & (0x0000ffff)],
		GPUMemSamplePos.x, GPUMemSamplePos.y, GPUMemSamplePos.z);
}

__device__ glm::u8vec4 rgbaFloatToUbyte4(float r, float g, float b, float a)
{
	r = __saturatef(r); // clamp to [0.0, 1.0]
	g = __saturatef(g);
	b = __saturatef(b);
	a = __saturatef(a);
	r *= 255.f;
	g *= 255.f;
	b *= 255.f;
	a *= 255.f;
	return glm::u8vec4(r, g, b, a);
}

__device__ void rayIntersectAABB(
	float* tEnter, float* tExit,
	const glm::vec3& rayOri, const glm::vec3& rayDrc,
	const glm::vec3& bot, const glm::vec3& top)
{
	// For  Ori + Drc * t3Bot.x = <Bot.x, 0, 0>
	// Thus t3Bot.x = Bot.x / Drc.x
	// Thus t3Bot.y = Bot.x / Drc.y
	// If  \
			//  \_\|\ 
			//   \_\|
			//      \.t3Bot.x
			//      |\
			//    __|_\.___|
			//      |  \t3Bot.y
			//    __|___\._|_
			//    t3Top.y\ |
			//      |     \.t3Top.x
			// 
			// Then t3Min = t3Bot, t3Max = t3Top
			// And  the max of t3Min is tEnter
			// And  the min of t3Max is tExit

	glm::vec3 invRay = 1.f / rayDrc;
	glm::vec3 t3Bot = invRay * (bot - rayOri);
	glm::vec3 t3Top = invRay * (top - rayOri);
	glm::vec3 t3Min{
		fminf(t3Bot.x, t3Top.x),
		fminf(t3Bot.y, t3Top.y),
		fminf(t3Bot.z, t3Top.z) };
	glm::vec3 t3Max{
		fmaxf(t3Bot.x, t3Top.x),
		fmaxf(t3Bot.y, t3Top.y),
		fmaxf(t3Bot.z, t3Top.z) };
	*tEnter = fmaxf(fmaxf(t3Min.x, t3Min.y), fmaxf(t3Min.x, t3Min.z));
	*tExit = fminf(fminf(t3Max.x, t3Max.y), fminf(t3Max.x, t3Max.z));
}

__device__ glm::vec3 phongShadingLOD0(
	const glm::vec3& rayDrc, const glm::vec3& samplePos,
	const glm::vec3& diffuseColor)
{
	glm::vec3 N;
	{
		float val1, val2;
		val1 = virtualSampleLOD0(samplePos + glm::vec3{ 1.f,0,0 });
		val2 = virtualSampleLOD0(samplePos - glm::vec3{ 1.f,0,0 });
		N.x = val2 - val1;
		val1 = virtualSampleLOD0(samplePos + glm::vec3{ 0,1.f,0 });
		val2 = virtualSampleLOD0(samplePos - glm::vec3{ 0,1.f,0 });
		N.y = val2 - val1;
		val1 = virtualSampleLOD0(samplePos + glm::vec3{ 0,0,1.f });
		val2 = virtualSampleLOD0(samplePos - glm::vec3{ 0,0,1.f });
		N.z = val2 - val1;
	}
	N = glm::normalize(N);

	glm::vec3 L = { -rayDrc.x,-rayDrc.y,-rayDrc.z };
	glm::vec3 R = L;
	if (glm::dot(N, L) < 0) N = -N;

	glm::vec3 ambient = d_renderParam.lightParam.ka * diffuseColor;
	glm::vec3 specular = glm::vec3(d_renderParam.lightParam.ks
		* powf(fmaxf(dot(N, (L + R) / 2.f), 0),
			d_renderParam.lightParam.shininess));
	glm::vec3 diffuse = d_renderParam.lightParam.kd
		* fmaxf(dot(N, L), 0.f) * diffuseColor;

	return ambient + specular + diffuse;
}

// WARNING:
// - Declaring type of param d_depth as [const hipTextureObject_t &]
//   will cause unknown error at tex2D()
__global__ void renderKernel(
	glm::u8vec4* d_colorL, glm::u8vec4* d_colorR,
	hipTextureObject_t d_depthTexL, hipTextureObject_t d_depthTexR)
{
	uint32_t windowX = blockIdx.x * blockDim.x + threadIdx.x;
	uint32_t windowY = blockIdx.y * blockDim.y + threadIdx.y;
	if (windowX >= d_renderParam.windowSize.x || windowY >= d_renderParam.windowSize.y) return;
	size_t windowFlatIdx = (size_t)windowY * d_renderParam.windowSize.x + windowX;

	// render Left or Right Eye
	glm::u8vec4& d_color = blockIdx.z == 0 ?
		d_colorL[windowFlatIdx] : d_colorR[windowFlatIdx];
	d_color = rgbaFloatToUbyte4(
		d_renderParam.lightParam.bkgrndColor.r,
		d_renderParam.lightParam.bkgrndColor.g,
		d_renderParam.lightParam.bkgrndColor.b,
		d_renderParam.lightParam.bkgrndColor.a);
#define TEST_SUBSAMPLE_SURFACE
#ifdef TEST_SUBSAMPLE_SURFACE
	float normX = (float)windowX / d_renderParam.windowSize.x;
	float normY = (float)windowY / d_renderParam.windowSize.y;
	float4 sbsmplVal = tex2D<float4>(d_sbsmplTexes[d_renderParam.sbsmplLvl - 1], normX, normY);
	d_color = rgbaFloatToUbyte4(sbsmplVal.x, sbsmplVal.y, sbsmplVal.z, 1.f);
	return;
#endif // TEST_SUBSAMPLE_SURFACE


	glm::vec3 rayDrc;
	const glm::vec3& camPos = d_renderParam.camPos2[blockIdx.z];
	const glm::mat4 unProjection = d_renderParam.unProjection2[blockIdx.z];
	float tEnter, tExit;
	{
		// find Ray of each Pixel on Window
		//   unproject
		glm::vec4 v41 = unProjection * glm::vec4{
			(((float)windowX / d_renderParam.windowSize.x) - .5f) * 2.f,
			(((float)windowY / d_renderParam.windowSize.y) - .5f) * 2.f,
			1.f, 1.f };
		//   don't rotate first to compute the Near&Far-clip steps
		rayDrc.x = v41.x, rayDrc.y = v41.y, rayDrc.z = v41.z;
		rayDrc = glm::normalize(rayDrc);
		float absRayDrcZ = fabsf(rayDrc.z);
		float tNearClip = d_renderParam.nearClip / absRayDrcZ;
		float tFarClip = d_renderParam.farClip;
		//   then compute upper bound of steps
		//   for Mesh-Volume mixed rendering
		{
			uchar4 depth4 = blockIdx.z == 0 ?
				tex2D<uchar4>(d_depthTexL, windowX, windowY) :
				tex2D<uchar4>(d_depthTexR, windowX, windowY);
			float meshBoundDep = d_renderParam.projection23 /
				((depth4.x / 255.f * 2.f - 1.f) + d_renderParam.projection22);
			if (tFarClip > meshBoundDep)
				tFarClip = meshBoundDep;
		} 
		tFarClip /= absRayDrcZ;
		//   rotate
		v41.x = rayDrc.x, v41.y = rayDrc.y, v41.z = rayDrc.z; // normalized in vec3
		v41 = d_renderParam.camRotaion * v41;
		rayDrc.x = v41.x, rayDrc.y = v41.y, rayDrc.z = v41.z;

		// Ray intersect Subregion(OBB)
		// equivalent to Ray intersect AABB in Subreion Space
		//   for pos, apply Rotation and Translation
		glm::vec4 v42{ camPos.x, camPos.y, camPos.z, 1.f };
		v42 = d_renderParam.subrgn.fromWorldToSubrgn * v42;
		//   for drc, apply Rotation only
		v41.w = 0;
		v41 = d_renderParam.subrgn.fromWorldToSubrgn * v41;
		rayIntersectAABB(
			&tEnter, &tExit,
			glm::vec3(v42),
			glm::normalize(glm::vec3(v41)),
			glm::zero<glm::vec3>(),
			glm::vec3{
				d_renderParam.subrgn.halfW * 2,
				d_renderParam.subrgn.halfH * 2,
				d_renderParam.subrgn.halfD * 2 });

		// Near&Far-clip
		if (tEnter < tNearClip) tEnter = tNearClip;
		if (tExit > tFarClip) tExit = tFarClip;
	}

	// no intersection
	if (tEnter >= tExit)
		return;
	glm::vec3 rayPos = camPos + tEnter * rayDrc;

#ifdef TEST_RAY_ENTER_POSITION
	// TEST: Ray Enter Position
	d_color = rgbaFloatToUbyte4(
		.5f * rayPos.x / d_renderParam.subrgn.halfW,
		.5f * rayPos.y / d_renderParam.subrgn.halfH,
		.5f * rayPos.z / d_renderParam.subrgn.halfD, 1.f);
	return;
#endif // TEST_RAY_ENTER_POSITION

#ifdef TEST_RAY_EXIT_POSITION
	// TEST: Ray Exit Position
	rayPos = camPos + tExit * rayDrc;
	d_color = rgbaFloatToUbyte4(
		.5f * rayPos.x / d_renderParam.subrgn.halfW,
		.5f * rayPos.y / d_renderParam.subrgn.halfH,
		.5f * rayPos.z / d_renderParam.subrgn.halfD, 1.f);
	return;
#endif // TEST_RAY_EXIT_POSITION

	glm::vec3 subrgnCenterInWdSp = {
		.5f * d_renderParam.subrgn.halfW,
		.5f * d_renderParam.subrgn.halfH,
		.5f * d_renderParam.subrgn.halfD,
	};
	glm::vec3 rayDrcMulStp = rayDrc * d_renderParam.step;
	glm::vec3 samplePos;
	glm::vec4 color = glm::zero<glm::vec4>();
	float sampleVal = 0;
	uint32_t stepNum = 0;
	for (;
		stepNum <= d_renderParam.maxStepNum && tEnter <= tExit;
		++stepNum, tEnter += d_renderParam.step, rayPos += rayDrcMulStp)
	{
		// ray pos in World Space -> sample pos in Voxel Space
		samplePos =
			(rayPos - subrgnCenterInWdSp + d_renderParam.subrgn.center)
			/ d_compVolumeParam.spaces;

		// virtual sample in Voxel Space, real sample in GPU Mem
		float currSampleVal = virtualSampleLOD0(samplePos);
		if (currSampleVal <= 0)
			continue;

		float4 currColor = tex2D<float4>(d_preIntTransferFunc, sampleVal, currSampleVal);
		if (currColor.w <= 0)
			continue;

		glm::vec3 shadingColor = phongShadingLOD0(rayDrc,
			samplePos, glm::vec3{ currColor.x,currColor.y,currColor.z });
		currColor.x = shadingColor.x;
		currColor.y = shadingColor.y;
		currColor.z = shadingColor.z;

		sampleVal = currSampleVal;
		color = color + (1.f - color.w) * glm::vec4{ currColor.x,currColor.y,currColor.z,currColor.w }
		*glm::vec4{ currColor.w,currColor.w,currColor.w,1.f };

		if (color.w > 0.9f)
			break;
	}

	// gamma correction
	constexpr float GAMMA_CORRECT_COEF = 1.f / 2.2f;
	color.r = powf(color.r, GAMMA_CORRECT_COEF);
	color.g = powf(color.g, GAMMA_CORRECT_COEF);
	color.b = powf(color.b, GAMMA_CORRECT_COEF);

	d_color = rgbaFloatToUbyte4(color.r, color.g, color.b, color.a);
}

void kouek::CompVolumeRendererCUDA::FAVRFunc::render(
	uint32_t windowW, uint32_t windowH, uint8_t sbsmplLvl)
{
	if (stream == nullptr)
		CUDA_RUNTIME_CHECK(hipStreamCreate(&stream));

	assert(sbsmplLvl > 0 && sbsmplLvl <= MAX_SUBSAMPLE_LEVEL_NUM);
	if (d_sbssmplTexArrs[sbsmplLvl - 1] == nullptr)
		createSubsampleAndReconsTexes(sbsmplLvl, windowW, windowH);

	// from here, called per frame, thus no CUDA_RUNTIME_API_CHECK
	for (uint8_t idx = 0; idx < 2; ++idx)
	{
		hipGraphicsMapResources(1, &outColorTexRsc2[idx], stream);
		hipGraphicsSubResourceGetMappedArray(&d_colorArr2[idx], outColorTexRsc2[idx], 0, 0);

		hipGraphicsMapResources(1, &inDepthTexRsc2[idx], stream);
		hipGraphicsSubResourceGetMappedArray(&d_depthArr2[idx], inDepthTexRsc2[idx], 0, 0);
		depthTexDesc.rscDesc.res.array.array = d_depthArr2[idx];
		hipCreateTextureObject(&d_depthTex2[idx], &depthTexDesc.rscDesc,
			&depthTexDesc.texDesc, nullptr);
	}

	dim3 threadPerBlock = { 16, 16 };
	dim3 blockPerGrid = { (windowW + threadPerBlock.x - 1) / threadPerBlock.x / sbsmplLvl,
						 (windowH + threadPerBlock.y - 1) / threadPerBlock.y, 2 };
	renderKernel <<< blockPerGrid, threadPerBlock, 0, stream >>> (
		d_color2[0], d_color2[1], d_depthTex2[0], d_depthTex2[1]);

	for (uint8_t idx = 0; idx < 2; ++idx)
	{
		hipMemcpyToArray(d_colorArr2[idx], 0, 0,
			d_color2[idx], d_colorSize, hipMemcpyDeviceToDevice);

		d_colorArr2[idx] = d_depthArr2[idx] = nullptr;
		hipGraphicsUnmapResources(1, &outColorTexRsc2[idx], stream);
		hipGraphicsUnmapResources(1, &inDepthTexRsc2[idx], stream);
	}
}
